#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include "timer.h"
#include "vector.h"

#include <hip/hip_runtime.h>

void cudaSetupDevice()
{
   int device_count = 0;
   hipGetDeviceCount(&device_count);
   if(device_count == 0) {
      std::fprintf(stderr, "Error: CUDA device not found\n");
      std::exit(EXIT_FAILURE);
   }

   int dev = 0;
   ASSERT_CUDA_SUCCESS( hipSetDevice(dev) );
}

template<typename T>
void CPUVectorAdd(const Vector<T> & v1, const Vector<T> & v2, Vector<T> & v3)
{
   for(auto i = 0L; i < v1.size(); ++i)
      v3[i] = v1[i] + v2[i];
}

template<typename T>
__global__ void GPUVectorAdd(long int N, const T* v1, const T* v2, T* v3)
{
   long int ind = blockIdx.x * blockDim.x + threadIdx.x;

   for(; ind < N; ind += blockDim.x*gridDim.x)
      v3[ind] = v1[ind] + v2[ind];
}

int main()
{
   cudaSetupDevice();

   {
      long int n = 1 << 10;
      Vector<float> v1(n), v2(n), v3(n);
      v1.rand();
      v2.rand();

      GPUVector<float> v1_gpu = ToDevice(v1);
      GPUVector<float> v2_gpu = ToDevice(v2);
      GPUVector<float> v3_gpu(n);

      CPUVectorAdd(v1, v2, v3);

      timer t_cross{};
      GPUVectorAdd<<< 512, 256 >>>(v1.size(), v1_gpu.data(), v2_gpu.data(), v3_gpu.data());
      ASSERT_CUDA_SUCCESS( hipDeviceSynchronize() );
      std::printf("GPUVectorAdd on GPU took: %.4e seconds\n\n", t_cross.wall_time());

      FromDevice(v3_gpu, v2);
      assert(within_tol_abs(v2, v3));
   }

   hipDeviceReset();
   return EXIT_SUCCESS;
}

